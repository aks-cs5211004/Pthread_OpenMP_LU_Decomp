
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void cuda_add(int a, int b, int *c){
	*c = a + b;
    printf("Hello from %d \n", threadIdx.x);
}

int main (){
	int a,b,c;
	int *dev_c;

	a=3;
	b=4;

	int driverVersion;
	int runtimeVersion;

	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("%d, %d\n",driverVersion, runtimeVersion);


	hipMalloc((void**)&dev_c, sizeof(int));
	hipError_t err1 = hipGetLastError();
	if(err1 != hipSuccess)
		printf("Error %s\n",hipGetErrorString(err1));
	
	cuda_add<<<1,256>>>(a,b,dev_c);
	hipError_t err2 = hipGetLastError();
	if(err2 != hipSuccess)
		printf("Error %s\n",hipGetErrorString(err2));
	
	hipDeviceSynchronize();
	hipError_t err3 = hipGetLastError();
	if(err3 != hipSuccess)
		printf("Error %s\n",hipGetErrorString(err3));
	
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	hipError_t err4 = hipGetLastError();
	if(err4 != hipSuccess)
		printf("Error %s\n",hipGetErrorString(err4));
	
	printf("%d + %d is %d\n",a,b,c);
	hipFree(dev_c);
	return 0;
}
